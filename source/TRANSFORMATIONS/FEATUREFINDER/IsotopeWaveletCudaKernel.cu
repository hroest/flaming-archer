#include "hip/hip_runtime.h"
// -*- Mode: C++; tab-width: 2; -*-
// vi: set ts=2:
//
// --------------------------------------------------------------------------
//                   OpenMS Mass Spectrometry Framework
// --------------------------------------------------------------------------
//  Copyright (C) 2003-2008 -- Oliver Kohlbacher, Knut Reinert
//
//  This library is free software; you can redistribute it and/or
//  modify it under the terms of the GNU Lesser General Public
//  License as published by the Free Software Foundation; either
//  version 2.1 of the License, or (at your option) any later version.
//
//  This library is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
//  Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public
//  License along with this library; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
//
// --------------------------------------------------------------------------
// $Maintainer: Rene Hussong$
// --------------------------------------------------------------------------


//**************************
//Uses the sorting code provided by Alan Kaatz
//**************************

#include <OpenMS/TRANSFORMATIONS/FEATUREFINDER/IsotopeWaveletCudaKernel.h>


#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <iomanip>

#include <assert.h>
#include <hip/hip_runtime.h>


// Shared memory sort kernel
#define ELEMENTS_SORT   512
#define THREADS_SORT    (ELEMENTS_SORT >> 2)
#define SORT_NUM        0xFFFFFE00

// Shared memory merge kernel
#define ELEMENTS_MERGE  1024
#define MERGE_NUM       0xFFFFFC00
#define THREADS_MERGE   (ELEMENTS_MERGE >> 2)

// Global memory merge kernel
#define ELEMENTS_GL     2
#define THREADS_GL      256

// Minimum size sortOnDevice() can handle
#define MIN_SORT_SIZE   ELEMENTS_SORT


#define MAX_BLOCKS_PER_GRID 65535


texture<float,1> trans_intensities_tex, pos_tex;
texture<int, 1> sorted_positions_indices_tex;

namespace OpenMS
{
	
	int checkCUDAError(const char *msg)
	{
			hipError_t err = hipGetLastError();
			if( hipSuccess != err) 
			{
					fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
					return (-1);
			};     
			return (0);       
	}

	
	__device__ float isotope_wavelet (float tz1, float mz)
	{
		float fac (-(LAMBDA_Q_0 + LAMBDA_Q_1*mz + LAMBDA_Q_2*mz*mz));
		fac += (tz1-1)*__log2f(-fac)*ONEOLOG2E - lgammaf(tz1);
			
		return (__sinf((tz1-1)*WAVELET_PERIODICITY) * __expf(fac));
	}



	__global__ void ConvolutionIsotopeWaveletKernel(float* signal_pos, float* signal_int, const int from_max_to_left, const int from_max_to_right, float* result, 
		const unsigned int charge, const int block_size, const float peak_cutoff_intercept, const float peak_cutoff_slope)
	{
		// the device-shared memory storing one data block
		// This is currently hard-coded to 256 points, since we require two 4B floats for each
		// data point, leading to 2kB per block.
		__shared__ float signal_pos_block[BLOCK_SIZE_MAX];
		__shared__ float signal_int_block[BLOCK_SIZE_MAX];

		// load the data from device memory to shared memory. 
		// to distribute the loads as evenly as possible over the threads, each thread loads
		// the data point it will later compute in the output. the first wavelet_length threads
		// will also load the padding to the left of the signal, the last wavelet_length ones will 
		// load the padding to the right
		
		// we will silently ignore the first wavelet_length points in the output; these have to be
		// zero-padded by the calling function. our data organization is as follows: each block computes
		// a part of the output that is block_size-2*wavelet_length points long. For the computation, we
		// require wavelet_length points on the left and on the right so we can put the wavelet on all
		// points even at the boundary.
			
		int problem_size_num_of_threads = block_size-(from_max_to_left+from_max_to_right/*+1-1*/);
		// the position in the original signal array that corresponds the data point computed by this thread
		//
		//                 left padding,                                                      position of thread
		//                 ignored in output    the points computed by the previous blocks    in block
		int my_data_pos  = from_max_to_left    +  blockIdx.x*problem_size_num_of_threads + threadIdx.x;
		int my_local_pos = threadIdx.x + from_max_to_left;

		//every thread loads its own (maximum) position into the shared memory
		signal_pos_block[my_local_pos] = signal_pos[my_data_pos];
		//signal_pos_block[my_local_pos] = tex1Dfetch(pos_tex, my_data_pos);
		signal_int_block[my_local_pos] = signal_int[my_data_pos];
		//printf ("%i\t\t%i\t\t %f\t\t%f\n",  blockIdx.x, threadIdx.x, signal_int[my_data_pos], tex1Dfetch(cuda_device_intens_texture_reference_, my_data_pos));
		//printf ("norm-loading: %f\t%i\n", signal_pos[my_data_pos], my_local_pos);

		//every thread with an ID smaller than the number of from_max_to_left loads the additional boundary points
		//at the left end
		if (threadIdx.x < from_max_to_left)
		{
			signal_pos_block[threadIdx.x] = signal_pos[my_data_pos-from_max_to_left];
			//signal_pos_block[threadIdx.x] = tex1Dfetch(pos_tex, my_data_pos-from_max_to_left);
			signal_int_block[threadIdx.x] = signal_int[my_data_pos-from_max_to_left];
			//printf ("pre-loading: %f\t%i\n",  signal_pos[my_data_pos-from_max_to_left], threadIdx.x);
			

			signal_pos_block[block_size-threadIdx.x-1] = signal_pos[my_data_pos+block_size-2*threadIdx.x-1-from_max_to_left];
			//signal_pos_block[block_size-threadIdx.x-1] = tex1Dfetch(pos_tex, my_data_pos+block_size-2*threadIdx.x-1-from_max_to_left);
			signal_int_block[block_size-threadIdx.x-1] = signal_int[my_data_pos+block_size-2*threadIdx.x-1-from_max_to_left];
			//printf ("extra-loading: %f\t%i\n", signal_pos[my_data_pos+block_size-2*threadIdx.x-1-from_max_to_left], block_size-threadIdx.x-1);

		}
		else
		{
			//int num_threads_with_only_one_load = problem_size_num_of_threads-from_max_to_left;
			//additional loads to be done at the right end: from_max_to_right
			int additional_right_end_loads=0;
			//printf ("crit: %i\t%i\t%i\n", (int)(from_max_to_right/num_threads_with_only_one_load), from_max_to_right, num_threads_with_only_one_load); 
			//while ((int)(from_max_to_right/num_threads_with_only_one_load) - additional_right_end_loads > 0)
			//while (my_local_pos + (additional_right_end_loads+1)*num_threads_with_only_one_load < block_size-2*from_max_to_left)
			while (my_local_pos + (additional_right_end_loads+1)*(problem_size_num_of_threads-from_max_to_left) < block_size-from_max_to_left)
			{
					++additional_right_end_loads;
					signal_pos_block[my_local_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)] = signal_pos[my_data_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)];
					//signal_pos_block[my_local_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)] = tex1Dfetch(pos_tex, my_data_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left));

					signal_int_block[my_local_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)] = signal_int[my_data_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)];
					//printf ("post-loading: %f\t%i\n", signal_pos[my_data_pos+additional_right_end_loads*num_threads_with_only_one_load], my_local_pos+additional_right_end_loads*num_threads_with_only_one_load);
			};
			/*additional_right_end_loads=0;		
			//leave this loops separated from each other; this reduces the number of registers in use, which can have significant impact on the occupancy of the program
			while (my_local_pos + (additional_right_end_loads+1)*(problem_size_num_of_threads-from_max_to_left) < block_size-from_max_to_left)
			{
					++additional_right_end_loads;
					//signal_pos_block[my_local_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)] = signal_pos[my_data_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)];
					signal_int_block[my_local_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)] = signal_int[my_data_pos+additional_right_end_loads*(problem_size_num_of_threads-from_max_to_left)];
					//printf ("post-loading: %f\t%i\n", signal_pos[my_data_pos+additional_right_end_loads*num_threads_with_only_one_load], my_local_pos+additional_right_end_loads*num_threads_with_only_one_load);
			};*/

		};
		//wait until the shared data is loaded completely
		__syncthreads(); 

		//exit(-1);

		//my_local_pos = threadIdx.x*(from_max_to_left+from_max_to_right+1) % problem_size_num_of_threads + from_max_to_left;
		//my_data_pos = my_local_pos + blockIdx.x*problem_size_num_of_threads;

		//printf ("%i\t%i\n", my_local_pos, my_data_pos); 

		float value = 0, boundary=(ceil(peak_cutoff_intercept+peak_cutoff_slope*charge*signal_pos_block[my_local_pos])*NEUTRON_MASS)/charge, c_diff;
		for (int current_conv_pos = my_local_pos-from_max_to_left; 
								current_conv_pos < my_local_pos+from_max_to_right; 
							++current_conv_pos)
		{
			//printf ("%i\t", current_conv_pos);
			//current_value = signal_int_block[current_conv_pos];
			//current_pos   = signal_pos_block[current_conv_pos];
			//if ((signal_pos_block[current_conv_pos]-center_pos) < 0) continue;
			//c_val = isotope_wavelet((current_pos-center_pos)*charge+1., current_pos);
			c_diff = signal_pos_block[current_conv_pos]-signal_pos_block[my_local_pos]+QUARTER_NEUTRON_MASS/charge;

			value += c_diff > 0 && c_diff <= boundary ? isotope_wavelet(c_diff*charge+1., signal_pos_block[current_conv_pos]*charge)*signal_int_block[current_conv_pos] : 0;
		};

		result[my_data_pos] = value;
	}


	void getExternalCudaTransforms (dim3 dimGrid, dim3 dimBlock, float* positions_dev, float* intensities_dev, int from_max_to_left, int from_max_to_right, float* result_dev, 
		const int charge, const int block_size, const float peak_cutoff_intercept, const float peak_cutoff_slope) 
	{
		ConvolutionIsotopeWaveletKernel<<<dimGrid,dimBlock>>> (positions_dev, intensities_dev, from_max_to_left, from_max_to_right, result_dev, charge, block_size, peak_cutoff_intercept, peak_cutoff_slope);
		hipDeviceSynchronize();
		checkCUDAError("ConvolutionIsotopeWaveletKernel");
	}

	__device__ inline void swap(float &a, float &b, int &c, int &d) 
	{
			float tmp (a);
			a = b;
			b = tmp;
				
			int tmp2 (c);
			c = d;
			d = tmp2;
	}

	__global__ void sharedMemMerge(float *array, int *pos, int k) {

			__shared__ float shmem[ELEMENTS_MERGE];
			__shared__ int posshmem[ELEMENTS_MERGE];

			int tmp = blockIdx.x * ELEMENTS_MERGE + threadIdx.x;

			float data = array[tmp];
			float data2 = array[tmp + (ELEMENTS_MERGE / 2)];

			float data3 = array[tmp + THREADS_MERGE];
			float data4 = array[tmp + THREADS_MERGE + (ELEMENTS_MERGE / 2)];
			
			int posdata = pos[tmp];
			int posdata2 = pos[tmp + (ELEMENTS_MERGE / 2)];

			int posdata3 = pos[tmp + THREADS_MERGE];
			int posdata4 = pos[tmp + THREADS_MERGE + (ELEMENTS_MERGE / 2)];

			int dir = k & (blockIdx.x * (ELEMENTS_MERGE));


			if (dir == 0) {
					if (data > data2) {  // ascending
							shmem[threadIdx.x] = data2;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = data;
							posshmem[threadIdx.x] = posdata2;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = posdata;
					} else {
							shmem[threadIdx.x] = data;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = data2;
							posshmem[threadIdx.x] = posdata;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = posdata2;
					}

					if (data3 > data4) {  // ascending
							shmem[threadIdx.x + THREADS_MERGE] = data4;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = data3;
							posshmem[threadIdx.x + THREADS_MERGE] = posdata4;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = posdata3;
					} else {
							shmem[threadIdx.x + THREADS_MERGE] = data3;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = data4;
							posshmem[threadIdx.x + THREADS_MERGE] = posdata3;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = posdata4;
					}
			} else {
					if (data < data2) {  // descending
							shmem[threadIdx.x] = data2;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = data;
							posshmem[threadIdx.x] = posdata2;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = posdata;
					} else {
							shmem[threadIdx.x] = data;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = data2;							
							posshmem[threadIdx.x] = posdata;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2)] = posdata2;
					}

					if (data3 < data4) {  // descending
							shmem[threadIdx.x + THREADS_MERGE] = data4;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = data3;
							posshmem[threadIdx.x + THREADS_MERGE] = posdata4;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = posdata3;
					} else {
							shmem[threadIdx.x + THREADS_MERGE] = data3;
							shmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = data4;							
							posshmem[threadIdx.x + THREADS_MERGE] = posdata3;
							posshmem[threadIdx.x + (ELEMENTS_MERGE / 2) + THREADS_MERGE] = posdata4;
					}
			}



			int j = 256, s = MERGE_NUM >> 2; 


			int x = threadIdx.x + (s & threadIdx.x);
			int y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}
			
			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);	
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + THREADS_MERGE) + ((threadIdx.x + THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}


			__syncthreads();

			int i = blockIdx.x * ELEMENTS_MERGE + threadIdx.x;
			array[i] = shmem[threadIdx.x];
			array[i + (ELEMENTS_MERGE / 4)] = shmem[(ELEMENTS_MERGE / 4) + threadIdx.x];
			array[i + (ELEMENTS_MERGE / 2)] = shmem[(ELEMENTS_MERGE / 2) + threadIdx.x];
			array[i + (3 * ELEMENTS_MERGE / 4)] = shmem[(3 * ELEMENTS_MERGE / 4) + threadIdx.x];

			pos[i] = posshmem[threadIdx.x];
			pos[i + (ELEMENTS_MERGE / 4)] = posshmem[(ELEMENTS_MERGE / 4) + threadIdx.x];
			pos[i + (ELEMENTS_MERGE / 2)] = posshmem[(ELEMENTS_MERGE / 2) + threadIdx.x];
			pos[i + (3 * ELEMENTS_MERGE / 4)] = posshmem[(3 * ELEMENTS_MERGE / 4) + threadIdx.x];
	}




	__global__ void mergeArray(float *array, int* pos, int j, int k, int s) {
			int tmp = (blockIdx.x * THREADS_GL);
			int x = tmp +  threadIdx.x + (tmp & s);
			j += x;

			float data1 = array[x];
			float data2 = array[j];

			if ((x & k) == 0) {    // ascending
					if (data1 > data2) {
							swap(array[x], array[j], pos[x], pos[j]);
					}
			} else {                // descending
					if (data1 < data2) {
							swap(array[x], array[j], pos[x], pos[j]);
					}
			}
	}




	__global__ void sharedMemSort(float2 *array, int2 *pos) 
	{
			__shared__ float shmem[ELEMENTS_SORT];
			__shared__ int posshmem[ELEMENTS_SORT];

			float2 data = array[blockIdx.x * (ELEMENTS_SORT / 2) + threadIdx.x];
			int2 posdata = pos[blockIdx.x * (ELEMENTS_SORT / 2) + threadIdx.x];


			if ( (threadIdx.x & 1) == 0) {
					if (data.x > data.y) {  // ascending
							shmem[2 * threadIdx.x] = data.y;
							shmem[2 * threadIdx.x + 1] = data.x;
							posshmem[2 * threadIdx.x] = posdata.y;
							posshmem[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem[2 * threadIdx.x] = data.x;
							shmem[2 * threadIdx.x + 1] = data.y;									
							posshmem[2 * threadIdx.x] = posdata.x;
							posshmem[2 * threadIdx.x + 1] = posdata.y;							
					}
			} else {
					if (data.x < data.y) {  // descending
							shmem[2 * threadIdx.x] = data.y;
							shmem[2 * threadIdx.x + 1] = data.x;							
							posshmem[2 * threadIdx.x] = posdata.y;
							posshmem[2 * threadIdx.x + 1] = posdata.x;

					} else {
							shmem[2 * threadIdx.x] = data.x;
							shmem[2 * threadIdx.x + 1] = data.y;							
							posshmem[2 * threadIdx.x] = posdata.x;
							posshmem[2 * threadIdx.x + 1] = posdata.y;
					}
			}


			for (int k = 4, r = 0xFFFFFFFC; k <= (ELEMENTS_SORT / 2); k *= 2, r <<= 1) {

					for (int j = k >> 1, s = r >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							
							__syncthreads();
							
							if ((x & k) == 0) {

									if (shmem[x] > shmem[y]) {  // ascending
											swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
									}
							} else {
									if (shmem[x] < shmem[y]) {  // descending
											swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
									}
							}
					}
			}


			data = array[blockIdx.x * (ELEMENTS_SORT / 2) + THREADS_SORT + threadIdx.x];
			posdata = pos[blockIdx.x * (ELEMENTS_SORT / 2) + THREADS_SORT + threadIdx.x];
			float* shmem2 = &shmem[ELEMENTS_SORT / 2];
			int* posshmem2 = &posshmem[ELEMENTS_SORT / 2];
			
			__syncthreads();

			if ( (threadIdx.x & 1) == 0) {
					if (data.x > data.y) {  // ascending
							shmem2[2 * threadIdx.x] = data.y;
							shmem2[2 * threadIdx.x + 1] = data.x;
							posshmem2[2 * threadIdx.x] = posdata.y;
							posshmem2[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem2[2 * threadIdx.x] = data.x;
							shmem2[2 * threadIdx.x + 1] = data.y;							
							posshmem2[2 * threadIdx.x] = posdata.x;
							posshmem2[2 * threadIdx.x + 1] = posdata.y;
					}
			} else {
					if (data.x < data.y) {  // descending
							shmem2[2 * threadIdx.x] = data.y;
							shmem2[2 * threadIdx.x + 1] = data.x;
							posshmem2[2 * threadIdx.x] = posdata.y;
							posshmem2[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem2[2 * threadIdx.x] = data.x;
							shmem2[2 * threadIdx.x + 1] = data.y;							
							posshmem2[2 * threadIdx.x] = posdata.x;
							posshmem2[2 * threadIdx.x + 1] = posdata.y;
					}
			}


			for (int k = 4, r = 0xFFFFFFFC; k <= (ELEMENTS_SORT / 2); k *= 2, r <<= 1) {

					for (int j = k >> 1, s = r >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if ((x & k) == 0) {
									if (shmem2[x] < shmem2[y]) {  // descending
											swap(shmem2[x], shmem2[y], posshmem2[x], posshmem2[y]);	
									}
							} else {
									if (shmem2[x] > shmem2[y]) {  // ascending
											swap(shmem2[x], shmem2[y], posshmem2[x], posshmem2[y]);
									}
							}
					}
			}


			if ((blockIdx.x & 1) == 0) {

					for (int j = ELEMENTS_SORT / 2, s = SORT_NUM >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if (shmem[x] > shmem[y]) {  // ascending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}

							x = (threadIdx.x + THREADS_SORT) + ((threadIdx.x + THREADS_SORT) & s);
							y = x + j;
			
							if (shmem[x] > shmem[y]) {  // ascending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}
					}
			
			} else {
			
					for (int j = ELEMENTS_SORT / 2, s = SORT_NUM >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if (shmem[x] < shmem[y]) {  // descending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}

							x = (threadIdx.x + THREADS_SORT) + ((threadIdx.x + THREADS_SORT) & s);
							y = x + j;
			
							if (shmem[x] < shmem[y]) {  // descending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}
					}
			
			}


			__syncthreads();

			int i = blockIdx.x * ELEMENTS_SORT + threadIdx.x;
			((float*)array)[i] = shmem[threadIdx.x];
			((float*)array)[i + (ELEMENTS_SORT / 4)] = shmem[(ELEMENTS_SORT / 4) + threadIdx.x];
			((float*)array)[i + (ELEMENTS_SORT / 2)] = shmem[(ELEMENTS_SORT / 2) + threadIdx.x];
			((float*)array)[i + (3 * ELEMENTS_SORT / 4)] = shmem[(3 * ELEMENTS_SORT / 4) + threadIdx.x];

			((int*)pos)[i] = posshmem[threadIdx.x];
			((int*)pos)[i + (ELEMENTS_SORT / 4)] = posshmem[(ELEMENTS_SORT / 4) + threadIdx.x];
			((int*)pos)[i + (ELEMENTS_SORT / 2)] = posshmem[(ELEMENTS_SORT / 2) + threadIdx.x];
			((int*)pos)[i + (3 * ELEMENTS_SORT / 4)] = posshmem[(3 * ELEMENTS_SORT / 4) + threadIdx.x];
	}


	__global__ void findCutOffIndex (float* array, int* cut_off_index)
	{
		int my_index = blockIdx.x*blockDim.x + threadIdx.x;
		if (my_index+1 >= gridDim.x*blockDim.x)
		{
			//printf ("returning\n");
			return;
		};
		float first = array[my_index], second = array[my_index+1];
		//printf ("my_index: %i\t%f\t%f\n", my_index, first, second);

		if (first <=0 && second > 0)
		{
			//printf ("writing: %i",  my_index+1);
			*cut_off_index = my_index+1;
		};
	};


	int sortOnDevice(float *array, int* pos_indices, int numElements, int padding)
	{
	  dim3 dimGridSharedMemSort((numElements / ELEMENTS_SORT) - (padding / ELEMENTS_SORT), 1, 1);
    dim3 dimBlockSharedMemSort(THREADS_SORT, 1, 1);

    dim3 dimGridMergeArray(numElements / (THREADS_GL * ELEMENTS_GL), 1, 1);
    dim3 dimBlockMergeArray(THREADS_GL, 1, 1);

    dim3 dimGridSharedMemMerge(numElements / ELEMENTS_MERGE, 1, 1);
    dim3 dimBlockSharedMemMerge(THREADS_MERGE, 1, 1);

		sharedMemSort<<<dimGridSharedMemSort, dimBlockSharedMemSort>>>(((float2*)array) + ((padding / ELEMENTS_SORT) * (ELEMENTS_SORT / 2)), ((int2*)pos_indices) + ((padding / ELEMENTS_SORT) * (ELEMENTS_SORT / 2)));

    for (int k = (ELEMENTS_SORT << 1), r = (int)(SORT_NUM << 1); k <= numElements; k *= 2, r <<= 1) 
		{

        for (int j = k / 2, s = r >> 1; j > (ELEMENTS_MERGE / 2); j >>= 1, s >>= 1) 
				{
            mergeArray<<<dimGridMergeArray, dimBlockMergeArray>>>(array, pos_indices, j, k, s);
        }

        sharedMemMerge<<<dimGridSharedMemMerge, dimBlockSharedMemMerge>>>(array, pos_indices, k);
    }		
		hipDeviceSynchronize();
		checkCUDAError("sortOnDevice");

		int num_threads = BLOCK_SIZE_MAX;
		while (numElements < num_threads && num_threads > 1)
		{
			num_threads /= 2;
		};

		if (num_threads == 1) //this case should never happen
		{
			return (0);
		}; 

		dim3 dimGrid (numElements/num_threads);
		dim3 dimBlock (num_threads);

		void* dev_cut_off_index;
		hipMalloc (&dev_cut_off_index, sizeof(int));
		hipMemset (dev_cut_off_index, -1, sizeof(int));

		//std::cout << "numElements:" << numElements << "\t" << dimGrid.x << "\t" << dimBlock.x << std::endl;

		findCutOffIndex<<<dimGrid, dimBlock>>> (array, (int*)dev_cut_off_index);
		hipDeviceSynchronize();
		checkCUDAError("findCutoffIndex");
		int cut_off_index=-1;
		hipMemcpy (&cut_off_index, dev_cut_off_index, sizeof(int), hipMemcpyDeviceToHost);

		//std::cout << "Found cutoff index as: " << cut_off_index << std::endl;
		return (cut_off_index);
	}


	

	extern __shared__ float c_scores [];
	__global__ void scoreIndividuals (int* sorted_positions_indices, float* pos, float* trans_intensities, float* scores, const int overall_size, 
		const int c, const int offset,  const int write_offset, const float peak_cutoff_intercept, const float peak_cutoff_slope)
	{		
		int v = threadIdx.x;
		//int ref_index = sorted_positions_indices[blockIdx.x+offset];
		int ref_index = tex1Dfetch (sorted_positions_indices_tex, blockIdx.x+offset);

		//printf ("my_index: %i\n", my_index);
		//printf ("ref_index: %i\n", ref_index); 	
	
		__shared__ int peak_cutoff, optimal_block_dim;
		__shared__ float seed_mz;

		if (v==0)
		{	
			seed_mz = tex1Dfetch(pos_tex, ref_index);
			peak_cutoff = (int) ceil(peak_cutoff_intercept+peak_cutoff_slope*seed_mz*(c+1));	
			optimal_block_dim = 4*(peak_cutoff-1) -1;
		};

		__syncthreads();
		if (v < optimal_block_dim)
		{
			float my_mz, l_pos, l_intens; int l_index;
			my_mz = seed_mz-((peak_cutoff-1)*NEUTRON_MASS-(v+1)*HALF_NEUTRON_MASS)/((float)(c+1));

			l_index = ref_index;
			if (my_mz > seed_mz)
			{ 
				while (l_index < overall_size && tex1Dfetch(pos_tex, l_index++) < my_mz) 
				{ 
				};
				l_index -= 2;
			}
			else
			{
				while (l_index >= 0 && tex1Dfetch(pos_tex,l_index--) > my_mz) 
				{							
				};
				++l_index;					
			};

			if (l_index >=0  && l_index+1 < overall_size)
			{	
				//l_pos = pos[l_index];
				l_pos = tex1Dfetch(pos_tex, l_index);
				//l_intens = trans_intensities[l_index];
				l_intens = tex1Dfetch(trans_intensities_tex, l_index);
				//c_scores[v] = l_intens + ( trans_intensities[l_index+1]-l_intens ) / (pos[l_index+1] - l_pos) * (my_mz - l_pos); 
				
				c_scores[v] = l_intens + ( tex1Dfetch(trans_intensities_tex, l_index+1)-l_intens ) / (tex1Dfetch(pos_tex, l_index+1) - l_pos) * (my_mz - l_pos); 				
				//printf ("Scoring: %f\t\t%f\t%f\t%f\t%f\t%f\n",  seed_mz, my_mz, tex1Dfetch(pos_tex, l_index+1), l_pos, tex1Dfetch(trans_intensities_tex, l_index+1), l_intens);
			}
			else
			{
				c_scores[v]=0;
			};
		};

		__syncthreads();	

		if (v==0)
		{
			float final_score = 0;
			int minus = -1;
			for (int i=0; i<optimal_block_dim; ++i)
			{
				final_score += minus*c_scores[i];
				minus *=-1;
			};
			scores[blockIdx.x+write_offset] = final_score;
		};			
	};


	void scoreOnDevice (int* sorted_positions_indices, float* trans_intensities, float* pos, float* scores, 
		const int c, const int num_of_scores, const int overall_size, const float peak_cutoff_intercept, const float peak_cutoff_slope, const unsigned int max_peak_cutoff)
	{
		//just to be sure, we will have 4 additional threads for numerical reasons that might trigger some additional scoring points
		dim3 blockDim (4*(max_peak_cutoff) -1); //the number of scoring points per candidates

		hipBindTexture(0, trans_intensities_tex, trans_intensities, overall_size*sizeof(float));
		hipBindTexture(0, pos_tex, pos, overall_size*sizeof(float));
		hipBindTexture(0, sorted_positions_indices_tex, sorted_positions_indices, overall_size*sizeof(int));	
		size_t schrott = overall_size - num_of_scores;

		dim3 gridDim;
		int counts=0, c_size = num_of_scores;
		while ((c_size -= MAX_BLOCKS_PER_GRID) > 0)
		{		
			gridDim = dim3 (MAX_BLOCKS_PER_GRID);

			scoreIndividuals<<<gridDim, blockDim, blockDim.x*sizeof(float)>>> (sorted_positions_indices, pos, trans_intensities, scores, overall_size, c, counts*MAX_BLOCKS_PER_GRID+schrott, counts*MAX_BLOCKS_PER_GRID, 
				peak_cutoff_intercept, peak_cutoff_slope);	
			++counts;
		};


		if ((c_size += MAX_BLOCKS_PER_GRID) > 0)
		{
			gridDim = dim3 (c_size);

			scoreIndividuals<<<gridDim, blockDim, blockDim.x*sizeof(float)>>> (sorted_positions_indices, pos, trans_intensities, scores, overall_size, c, counts*MAX_BLOCKS_PER_GRID+schrott, counts*MAX_BLOCKS_PER_GRID, 
				peak_cutoff_intercept, peak_cutoff_slope);		
		};
		
//		CUDA_SAFE_CALL(hipDeviceSynchronize());
		hipDeviceSynchronize();
		checkCUDAError("scoreOnDevice");
		
		hipUnbindTexture (trans_intensities_tex);
		hipUnbindTexture (pos_tex);
		hipUnbindTexture (sorted_positions_indices_tex);
	}

}	
